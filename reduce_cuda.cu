#include "hip/hip_runtime.h"
#include <thread>
#include "reduce.hpp"
#include "utils.hpp"
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <sys/time.h>

__global__ void reduce_block(float* g_idata, int reduce_size, float* g_odata)
{
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int start = blockIdx.x * reduce_size;

    sdata[tid] = 0;
    for (unsigned int i = tid; i < reduce_size; i += blockDim.x)
    {
        sdata[tid] += g_idata[i + start];
    }
    __syncthreads();

    // do reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

void reduce_cuda(const std::vector<float>& vec, int reduce_size, std::vector<float>& vec_out)
{
    //std::cout << "vec_cuda = " << std::endl;
    //std::cout << vec << std::endl;
    int elem_num = vec.size();
    int block_num = elem_num / reduce_size;
    vec_out.resize(block_num);

    float *dev_idata, *dev_odata;
    hipMalloc((void**)&dev_idata, elem_num * sizeof(float));
    hipMalloc((void**)&dev_odata, block_num * sizeof(float));
    hipMemcpy(dev_idata, vec.data(), elem_num * sizeof(float), hipMemcpyHostToDevice);
  
    int block_size = 128;

    timeval t1, t2;
    gettimeofday(&t1, nullptr);
    reduce_block<<<block_num, block_size, block_size * sizeof(float)>>>(dev_idata, reduce_size, dev_odata);
    hipDeviceSynchronize();
    gettimeofday(&t2, nullptr);

    float ms = (t2.tv_sec - t1.tv_sec) * 1000.0f + (t2.tv_usec - t1.tv_usec) / 1000.0f;
    std::cout << "Kernel takes " << ms << " ms" << std::endl;
    hipMemcpy(vec_out.data(), dev_odata, block_num * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_idata);
    hipFree(dev_odata);
}

